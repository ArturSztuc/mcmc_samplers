// std
#include <stdio.h>

// CUDA specifics
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
//#include <helper_cuda.h>

// Debugging macros
#define CUDA_ERROR_CHECK
#define CudaCheckError()  __cudaCheckError(__FILE__, __LINE__)


// Check if there's been an error
inline void __cudaCheckError( const char *file, const int line ) {
  // **************************************************
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
    exit(-1);
  }
#endif
  return;
}

__global__ void CalculateAutocorrelations(
    float * gParameterValues,
    float * gMeansGPU,
    int gParSizeAll,
    int gParSize,
    int gMaxLag,
    float * gAutocorrelations
    )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int bdim  = blockDim.x * gridDim.x;

  // Length of autocorrelation array
  int gParLag = gParSize * gMaxLag;
  // Number of MCMC steps
  int nSteps  = gParSizeAll/gParSize;

  // Is there a way to make this a matrix multiplication using Eigen?
  // parameter x step
  // parameter x lag

  for(int thread = index; thread < gParLag; thread += bdim)
  {
    int iPar = (thread / gMaxLag) % gParSize;
    int iLag = thread % gMaxLag;

    float numerator = 0;
    float denominator = 0;
    for(int i = 0; i < nSteps; ++i){
      int idxtmp = (iPar * nSteps) + i;
      float diff = gParameterValues[idxtmp] - gMeansGPU[iPar];

      if(i < nSteps - iLag){
        float lagTerm = gParameterValues[idxtmp + iLag] - gMeansGPU[iPar];
        float product = diff*lagTerm;
        numerator += product;
      }
      denominator += diff * diff;
    }
    gAutocorrelations[(iPar * gMaxLag) + iLag] = numerator/denominator;
  }
}

// Allocate memory on GPU
__host__ void InitValuesGPU(
    float **gParameterValues,
    float **gAutocorrelations,
    float **gMeans,
    int gParSizeAll, 
    int gMaxLag,
    int gNPars)
{
  hipMalloc((void**) gParameterValues, gParSizeAll*sizeof(float));
  CudaCheckError();
  printf("Allocated %i parameter entries with total memory of %f Mb\n", gParSizeAll, (gParSizeAll*sizeof(float))/1.E6);

  hipMalloc((void**) gAutocorrelations, gMaxLag*gNPars*sizeof(float));
  CudaCheckError();
  printf("Allocated %i autocorrelation entries with total memory of %f Mb\n", gMaxLag*gNPars, (gMaxLag*gNPars*sizeof(float))/1.E6);

  hipMalloc((void**) gMeans, gNPars*sizeof(float));
  CudaCheckError();
  printf("Allocated %i means with total memory of %f Mb\n", gNPars, (gNPars*sizeof(float))/1.E6);
}

// Copies CPU RAM to GPU
__host__ void CopyToGPU(
    float *gParameterValues,
    float *fParameterValues,
    float *gMeans,
    float *fMeans,
    int fParSizeAll,
    int fParSize
    )
 
{
  hipMemcpy(gParameterValues, fParameterValues, fParSizeAll*sizeof(float), hipMemcpyHostToDevice);
  CudaCheckError();
  printf("Copied parameter values to the GPU\n");

  hipMemcpy(gMeans, fMeans, fParSize*sizeof(float), hipMemcpyHostToDevice);
  CudaCheckError();
  printf("Copied parameter means to the GPU\n");
}

__host__ void RunAutocorrelationsGPU(
      float *gParameterValues,
      float *gAutocorrelations,
      float *gMeansGPU,
      int gParSizeAll, 
      int gParSize,
      int gMaxLag,
      float *fAutocorrelations)
{

  int block_size = 1024;
  int num_blocks = ((gMaxLag * gParSize) + block_size - 1) / block_size;

  printf("Calculating Autocorrelations...\n");
  CalculateAutocorrelations<<<num_blocks, block_size>>>(
      gParameterValues,
      gMeansGPU,
      gParSizeAll,
      gParSize,
      gMaxLag,
      gAutocorrelations
      );
  CudaCheckError();

  printf("Copying Autocorrelations from the GPU back to Host RAM...\n");
  hipMemcpy(fAutocorrelations, gAutocorrelations, gParSize * gMaxLag*sizeof(float), hipMemcpyDeviceToHost);
  CudaCheckError();
}

__host__ void ClearValuesGPU(
    float *gParameterValues,
    float *gAutocorrelations,
    float *gMeansGPU)
{
  hipFree(gParameterValues);
  hipFree(gAutocorrelations);
  hipFree(gMeansGPU);
}
